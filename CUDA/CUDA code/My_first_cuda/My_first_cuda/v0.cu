
#include <hip/hip_runtime.h>
/*
#include <stdio.h>
#include<cmath>
#include <cuda.h>
#include "cuda_runtime.h"
#include<iostream>
using namespace std;

//latency: 3.835ms
template<int blockSize>
__global__ void reduce_v0(float* d_in, float* d_out) {
    __shared__ float smem[blockSize];

    int tid = threadIdx.x;
    // int gtid = blockIdx.x * blockSize + threadIdx.x;
    // load: ÿ���̼߳���һ��Ԫ�ص�shared mem��Ӧλ��
    smem[tid] = d_in[tid];
    __syncthreads();

    // compute: reduce in shared mem
    // ˼����������β��е�
    for (int index = 1; index < blockDim.x; index *= 2) {
        if (tid % (2 * index) == 0) {
            smem[tid] += smem[tid + index];
        }
        __syncthreads();
    }

    // store: write back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}
bool CheckResult(float* out, float groudtruth, int n) {
    float res = 0;
    for (int i = 0; i < n; i++) {
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main() {
    float milliseconds = 0;
    const int N = 25600000;
    cudaSetDevice(0);
    cudaDeviceProp deviceProp;
    cudaGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float* a = (float*)malloc(N * sizeof(float));
    float* d_a;
    cudaMalloc((void**)&d_a, N * sizeof(float));

    float* out = (float*)malloc((GridSize) * sizeof(float));
    float* d_out;
    cudaMalloc((void**)&d_out, (GridSize) * sizeof(float));

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    cudaMemcpy(d_a, a, N * sizeof(float), cudaMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    cudaEventRecord(start);
    reduce_v0<blockSize> << <Grid, Block >> > (d_a, d_out);
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&milliseconds, start, stop);

    cudaMemcpy(out, d_out, GridSize * sizeof(float), cudaMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if (is_right) {
        printf("the ans is right\n");
    }
    else {
        printf("the ans is wrong\n");
        //for(int i = 0; i < GridSize;i++){
            //printf("res per block : %lf ",out[i]);
        //}
        //printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v0 latency = %f ms\n", milliseconds);

    cudaFree(d_a);
    cudaFree(d_out);
    free(a);
    free(out);
}*/