#include "hip/hip_runtime.h"
#include"10.fused_bias_mask_scale_and_add.h"
# include<cstdint>// uint8_t
# include<iostream>
#include<hip/hip_fp16.h>
# include<cassert>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"
#include"utils.h"




template<typename T>
struct MaskAndNormFunctor {
	// mask
	const uint8_t* mask;
	// Norm calculate
	const T* add_val;
	float scale;
	MaskAndNormFunctor(const uint8_t* mask, const T* add_val, float scale)
		:mask(mask), add_val(add_val),scale(scale){}

	__device__ T Compute(T x, int64_t i) const {
		return x * static_cast<T>(static_cast<bool>(mask[i])*scale) + add_val[i];
	}


};
template<typename FUNCTOR, typename T>
__global__ void FusedBiasAddCUDAKernelFloat(
	FUNCTOR functor,
	const int elem_cnt,
	const int bias_size,
	const T* x,
	const T* bias,
	T* y) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int KernelSize = blockDim.x * gridDim.x;
	for (int i = tid; i < elem_cnt; i += KernelSize) {
		// Add MLP bias
		T x_i = x[i] + bias[i % bias_size];
		y[i] = functor.Compute(x_i, i);
	}
}

template<typename T>
void CPU_fused_kernel(
	uint8_t* mask, 
	T* add_val, 
	float scale, 
	T* cpu_input,
	int ele_cnt,
	T* bias,
	int bias_size,
	T* output) 
{

	for (int i = 0; i < ele_cnt; i++) {
		T x_i = cpu_input[i] + bias[i % bias_size];
		output[i] = x_i * static_cast<T>(static_cast<bool>(mask[i]) * scale) + add_val[i];
	}

}
template<typename T>
bool CHECK_RES(T* cpu_output, T* gpu_output, const int ele_cnt) {
	for (int i = 0; i < ele_cnt; i++) {
		if (std::abs(cpu_output[i] - gpu_output[i]) > (T)(1e-5)) {
			std::cout << "��" << i << "��Ԫ�س���" << std::endl;
			std::cout << cpu_output[i] << " " << gpu_output[i] << std::endl;
			return false;
		}
	}
	std::cout << "����Ԫ�ض�һ��" << std::endl;
	return true;
}

void test_fp32_fused_kernel() {
	constexpr int ele_cnt = 1<<30;
	float scale = 0.5;

	// parameter in cpu
	uint8_t* mask_tensor = new uint8_t[ele_cnt];
	float* add_val = new float[ele_cnt];
	for (int i = 0; i < ele_cnt; i++) {
		mask_tensor[i] = (uint8_t)(i%2);
		add_val[i] = (float)(i%10) ;
	}

	// bias,input,output in cpu
	int bias_size = 1024;
	float* x = (float*)malloc(sizeof(float) * ele_cnt);
	float* y = (float*)malloc(sizeof(float) * ele_cnt);
	float* bias = (float*)malloc(sizeof(float) * bias_size);
	for (int i = 0; i < ele_cnt; i++) {
		x[i] = (float)(i%100);
	}
	for (int i = 0; i < bias_size; i++) {
		bias[i] = (float)(i%10);
	}

	// cpu_output
	float* cpu_output = (float*)malloc(sizeof(float) * ele_cnt);
	TIME_CPU(CPU_fused_kernel<float>(mask_tensor, add_val, scale, x, ele_cnt, bias, bias_size, cpu_output));


	// bias,input,output in gpu
	float* d_x, * d_y, * d_bias;
	hipMalloc((void**)&d_x, ele_cnt * sizeof(float));
	hipMalloc((void**)&d_y, ele_cnt * sizeof(float));
	hipMalloc((void**)&d_bias, bias_size * sizeof(float));

	hipMemcpy(d_x, x, sizeof(float) * ele_cnt, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(float) * ele_cnt, hipMemcpyHostToDevice);
	hipMemcpy(d_bias, bias, sizeof(float) * bias_size, hipMemcpyHostToDevice);

	// mask_tensor, add_cal in gpu
	uint8_t* d_mask_tensor;
	float* d_add_val;
	hipMalloc((void**)&d_mask_tensor, ele_cnt * sizeof(uint8_t));
	hipMalloc((void**)&d_add_val, ele_cnt* sizeof(float));
	hipMemcpy(d_mask_tensor, mask_tensor, sizeof(uint8_t) * ele_cnt, hipMemcpyHostToDevice);
	hipMemcpy(d_add_val, add_val, sizeof(float) * ele_cnt, hipMemcpyHostToDevice);

	/*
		����һ�� CUDA �ṩ�Ľṹ�壬���ڴ洢 GPU �豸�ĸ������ԡ�
		���а����� GPU �豸�����ơ������������ڴ��С������߳������������Ϳ��С����Ϣ��
	*/
	hipDeviceProp_t deviceProp;
	// ��ָ�� GPU �豸��������Ϣ��䵽 hipDeviceProp_t �ṹ���С�
	hipError_t message_GPU = hipGetDeviceProperties(&deviceProp, 0);

	int maxblocks = deviceProp.maxGridSize[0];
	
	int blockSize = 1024;
	int gridSize = std::min((ele_cnt + blockSize - 1) / blockSize, maxblocks);

	MaskAndNormFunctor<float> MNF(d_mask_tensor, d_add_val, scale);

	auto kernel_launcher = [&]() {
		FusedBiasAddCUDAKernelFloat << <gridSize, blockSize >> > (MNF,
			ele_cnt,
			bias_size,
			d_x,
			d_bias,
			d_y);
		};

	TIME_GPU(kernel_launcher);

	hipMemcpy(y, d_y, sizeof(float) * ele_cnt, hipMemcpyDeviceToHost);


	CHECK_RES(cpu_output, y, ele_cnt);

	delete[] mask_tensor;
	delete[] add_val;
	free(x);
	free(y);
	free(bias);
	free(cpu_output);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_bias);
	hipFree(d_mask_tensor);
	hipFree(d_add_val);
}



// fp16ƫ������
template<>
struct MaskAndNormFunctor<__half> {
	// mask
	const uint8_t* mask;
	// Norm calculate
	const __half* add_val;
	float scale;
	MaskAndNormFunctor(const uint8_t* mask, const __half* add_val, float scale)
		:mask(mask), add_val(add_val), scale(scale) {}

	// half compute
	__device__ __half Compute(__half x, int64_t i) const {
		return x * static_cast<half>(static_cast<bool>(mask[i]) * scale) + add_val[i];
	}

	//half2 compute
	__device__ __half2 VecCompute(__half2 x, int64_t i) const {

		// mask��uint8_tû�����������ݽṹ��ʹ�ÿ���һ��char��8λ��������Դ��棬���ǲ�����λ�õ�trickyд��
		const char2* mask_vec = reinterpret_cast<const char2*>(mask);

		const __half2* add_val_vec = reinterpret_cast<const __half2*> (add_val);

		char2 mask_val = mask_vec[i];//��������ȡ
		//ת��Ϊ__half������������
		__half2 one_or_zero;
		one_or_zero.x = mask_val.x;
		one_or_zero.y = mask_val.y;
		__half2 scale_vec = __float2half2_rn(scale);


		//__hmul2(x, one_or_zero), scale_vec)									: mask����
		//__hmul2(__hmul2(x, one_or_zero), scale_vec)							��Norm-scale����
		//__hadd2(__hmul2(__hmul2(x, one_or_zero), scale_vec), add_val_vec[i]);	��Norm-bias����
		return __hadd2(__hmul2(__hmul2(x, one_or_zero), scale_vec), add_val_vec[i]);
	}


};

template<typename FUNCTOR>
__global__ void FusedBiasAddCUDAKernelFloat<FUNCTOR,__half>(
	FUNCTOR functor,
	const int elem_cnt,
	const int bias_size,
	const __half* x,
	const __half* bias,
	__half* y)
{
	// pack
	const int h2_ele_cnt = elem_cnt / 2;
	//assert(elem_cnt % 2 == 0);
	const int h2_bias_size = bias_size / 2;
	//assert(elem_cnt % 2 == 0);
	const auto* x_h2 = reinterpret_cast<const __half2*>(x);
	const auto* bias_h2 = reinterpret_cast<const __half2*>(bias);
	auto* y_h2 = reinterpret_cast<__half2*>(y);


	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int kernelSize = blockDim.x * gridDim.x;
	for (int i = tid; i < h2_ele_cnt; i += kernelSize) {
		
		__half2 x_i = __hadd2(x_h2[i], bias_h2[i % h2_bias_size]);
		//y_h2[i] = functor.Compute(x_i, i);
		y_h2[i] = functor.VecCompute(x_i, i);
	}

}


void calculate_cpu_from_half(
	uint8_t* mask_tensor,
	__half* add_val,
	float scale,
	__half* x,
	int ele_cnt, 
	__half* bias,
	int bias_size, 
	float* cpu_output
	) {
	// convert
	float* fp_add_val = (float*)malloc(sizeof(float) * ele_cnt);
	float* fp_x = (float*)malloc(sizeof(float) * ele_cnt);
	float* fp_bias = (float*)malloc(sizeof(float) * ele_cnt);
	for (int i = 0; i < ele_cnt; i++) {
		fp_add_val[i] = __half2float(add_val[i]);
		fp_x[i] = __half2float(x[i]);
		
	}

	for (int i = 0; i < bias_size; i++) {
		fp_bias[i] = __half2float(bias[i]);

	}

	TIME_CPU(CPU_fused_kernel<float>(mask_tensor, fp_add_val, scale, fp_x, ele_cnt, fp_bias, bias_size, cpu_output));

	free(fp_add_val);
	free(fp_x);
	free(fp_bias);
}

float* convert2float(__half* gpu_result, int ele_cnt) {
	float* fp_gpu_result = (float*)malloc(sizeof(float) * ele_cnt);
	for (int i = 0; i < ele_cnt; i++) {
		fp_gpu_result[i] = __half2float(gpu_result[i]);
	}
	return fp_gpu_result;
}

void test_fp16_fused_kernel() {
	constexpr int ele_cnt = 1<<30;
	float scale = 0.5;

	// parameter in cpu
	uint8_t* mask_tensor = new uint8_t[ele_cnt];
	__half* add_val = new __half[ele_cnt];
	for (int i = 0; i < ele_cnt; i++) {
		mask_tensor[i] = (uint8_t)(i % 2);
		add_val[i] = (__half)(i%10);
	}

	// bias,input,output in cpu
	int bias_size = 1024;
	__half* x = (__half*)malloc(sizeof(__half) * ele_cnt);
	__half* y = (__half*)malloc(sizeof(__half) * ele_cnt);
	__half* bias = (__half*)malloc(sizeof(__half) * bias_size);
	for (int i = 0; i < ele_cnt; i++) {
		x[i] = (__half)(i%100);
	}
	for (int i = 0; i < bias_size; i++) {
		bias[i] = (__half)(i%10);
	}

	// cpu_output,ֻ��ʹ��float����
	
	float* cpu_output = (float*)malloc(sizeof(float) * ele_cnt);
	calculate_cpu_from_half(mask_tensor, add_val, scale, x, ele_cnt, bias, bias_size, cpu_output);
	
	// bias,input,output in gpu
	__half* d_x, * d_y, * d_bias;
	hipMalloc((void**)&d_x, ele_cnt * sizeof(__half));
	hipMalloc((void**)&d_y, ele_cnt * sizeof(__half));
	hipMalloc((void**)&d_bias, bias_size * sizeof(__half));

	hipMemcpy(d_x, x, sizeof(__half) * ele_cnt, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(__half) * ele_cnt, hipMemcpyHostToDevice);
	hipMemcpy(d_bias, bias, sizeof(__half) * bias_size, hipMemcpyHostToDevice);

	// mask_tensor, add_cal in gpu
	uint8_t* d_mask_tensor;
	__half* d_add_val;
	hipMalloc((void**)&d_mask_tensor, ele_cnt * sizeof(uint8_t));
	hipMalloc((void**)&d_add_val, ele_cnt * sizeof(__half));
	hipMemcpy(d_mask_tensor, mask_tensor, sizeof(uint8_t) * ele_cnt, hipMemcpyHostToDevice);
	hipMemcpy(d_add_val, add_val, sizeof(__half) * ele_cnt, hipMemcpyHostToDevice);

	/*
		����һ�� CUDA �ṩ�Ľṹ�壬���ڴ洢 GPU �豸�ĸ������ԡ�
		���а����� GPU �豸�����ơ������������ڴ��С������߳������������Ϳ��С����Ϣ��
	*/
	hipDeviceProp_t deviceProp;
	// ��ָ�� GPU �豸��������Ϣ��䵽 hipDeviceProp_t �ṹ���С�
	hipError_t message_GPU = hipGetDeviceProperties(&deviceProp, 0);

	int maxblocks = deviceProp.maxGridSize[0];

	int blockSize = 1024;
	int gridSize = std::min((ele_cnt/2 + blockSize - 1) / blockSize, maxblocks);

	MaskAndNormFunctor<__half> MNF(d_mask_tensor, d_add_val, scale);

	auto kernel_launcher = [&]() {
		FusedBiasAddCUDAKernelFloat << <gridSize, blockSize >> >(MNF,
			ele_cnt,
			bias_size,
			d_x,
			d_bias,
			d_y);
		};

	TIME_GPU(kernel_launcher);

	hipMemcpy(y, d_y, sizeof(__half) * ele_cnt, hipMemcpyDeviceToHost);
	
	float* fp_gpu_result = convert2float(y, ele_cnt);

	CHECK_RES(cpu_output, fp_gpu_result, ele_cnt);
	
	delete[] mask_tensor;
	delete[] add_val;
	free(x);
	free(y);
	free(bias);
	free(cpu_output);
	free(fp_gpu_result);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_bias);
	hipFree(d_mask_tensor);
	hipFree(d_add_val);
}