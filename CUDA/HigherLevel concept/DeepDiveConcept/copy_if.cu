#include "hip/hip_runtime.h"
# include "copy_if.h"
# include <iostream>
# include <hip/hip_runtime.h>
# include <hip/hip_runtime.h>

__global__ void origin_copy_if(int* res, const int* src, int* nres, int n) {
	// ���ַ����õ���res������ѭԭsrc�з��������������е�˳��
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int total_size = gridDim.x * blockDim.x;
	int loop = n/total_size + 1;

	for(int i = 0; i < loop; i++)
	{
		int true_idx = idx + total_size * i;
		if ( true_idx < n && src[true_idx] > 0) {
			res[atomicAdd(nres, 1)] = src[true_idx];
		}
	}

}

__host__ int cpu_copy_if(int* res, const int* src, int n) {

	int resnum = 0;
	for (int i = 0; i < n; i++) {
		if (src[i] > 0) {
			res[resnum++] = src[i];
		}
	}
	return resnum;
}

__global__ void blocklevel_copy_if(int* res, const int* src, int* nres, int n) {
	// block�ڵļ���
	__shared__ int count;

	int gridsize = gridDim.x * blockDim.x;
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pos;
	// ѭ������
	for (int i = idx; i < n; i+= gridsize) {
		//��ֹƵ������λ���Դ��src
		int data = src[i];

		// ��ÿ��block�У�ָ��һ���߳����ȳ�ʼ��count
		if (threadIdx.x == 0) {
			count = 0;
		}
		__syncthreads();

		// ÿ���߳��ж��Ƿ�����Ҫ��
		if (i < n && data > 0) {
			//����Ҫ���ʹ��shared_memory variable��¼,pos��¼��ƫ��
			pos = atomicAdd(&count, 1);
		}

		__syncthreads();

		// ȫ����ɺ���һ���߳������Լ�block����Ϣ������block���ϣ���ȡblock�����ƫ��
		// ����countʵ�����Ǻ����ò����ˣ�ֱ�Ӹ�����Ϊbias
		// block��˳����ȫȡ�������block��thread0�ܹ����ϵ�ʱ��
		if (threadIdx.x == 0) {
			count = atomicAdd(nres, count);
		}
		__syncthreads();
		//��ʱ�����߳̿��Ի�ȡ����ȫ�ֵ�Ψһ����
		if (i < n && data > 0) {
			res[count + pos] = data;
		}

		__syncthreads();



	}

}