# include"copy_if_test.h"
void check_result(int* device_num, int* device_result, int* cpu_num, int* cpu_result) {
	if (*device_num != *cpu_num) {
		std::cout << "Device result is something wrong with cpu_result" << std::endl;
		std::cout << "device_num = " << *device_num << std::endl;
		std::cout << "cpu_num = " << *cpu_num << std::endl;

	}
	else {
		std::cout << "Device result is as same as cpu_result" << std::endl;
		/*
		int num = 0;
		for (int i = 0; i < *device_num; i++) {
			if (device_result[i] != cpu_result[i]) {
				num++;
				std::cout << "i-th result is not same where GPU is " << device_result[i] << " and CPU is " << cpu_result[i] << std::endl;
			}
		}
		std::cout << "The total no-match result num is " << num << std::endl;
		*/
	}
}

int* generate_random_array() {
	int* src = new int[array_size];


	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> distrib(-100, 100);
	for (int i = 0; i < array_size; i++) {
		src[i] = distrib(gen);
		//std::cout << src[i] << " ";
	}
	return src;
}

void compare1(int* src) {
	/*
	CPU vs original GPU copy_if
	*/
	int* res = new int[array_size];

	int resnum;
	TIME_CPU(resnum = cpu_copy_if(res, src, array_size));


	int* host_result1 = (int*)malloc(array_size * sizeof(int));
	int* host_nres1 = (int*)malloc(sizeof(int));
	int* device_src;
	int* device_result1;
	int* nres1;
	CHECK(hipMalloc((void**)&device_src, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&device_result1, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&nres1, sizeof(int)));

	CHECK(hipMemcpy(device_src, src, array_size * sizeof(int), hipMemcpyHostToDevice));

	auto kernel_launcher = [&]() {
		origin_copy_if << < 512, 1024 >> > (device_result1, device_src, nres1, array_size);
		};

	TIME_GPU(kernel_launcher);

	CHECK(hipMemcpy(host_result1, device_result1, sizeof(int) * array_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_nres1, nres1, sizeof(int), hipMemcpyDeviceToHost));


	check_result(host_nres1, host_result1, &resnum, res);

	hipFree(device_src);
	hipFree(device_result1);
	hipFree(nres1);
	free(host_result1);
	free(host_nres1);
	free(res);
}

void compare2(int* src) {
	/*
	CPU vs original GPU copy_if
	*/

	int* res = new int[array_size];

	int resnum;
	TIME_CPU(resnum = cpu_copy_if(res, src, array_size));


	int* host_result1 = (int*)malloc(array_size * sizeof(int));
	int* host_nres1 = (int*)malloc(sizeof(int));
	int* device_src;
	int* device_result1;
	int* nres1;
	CHECK(hipMalloc((void**)&device_src, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&device_result1, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&nres1, sizeof(int)));

	CHECK(hipMemcpy(device_src, src, array_size * sizeof(int), hipMemcpyHostToDevice));

	auto kernel_launcher = [&]() {
		blocklevel_copy_if << < 512, 1024 >> > (device_result1, device_src, nres1, array_size);
		};

	TIME_GPU(kernel_launcher);

	CHECK(hipMemcpy(host_result1, device_result1, sizeof(int) * array_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_nres1, nres1, sizeof(int), hipMemcpyDeviceToHost));


	check_result(host_nres1, host_result1, &resnum, res);

	hipFree(device_src);
	hipFree(device_result1);
	hipFree(nres1);
	free(host_result1);
	free(host_nres1);
	free(res);
}

void compare3(int* src) {
	/*
	CPU vs original GPU copy_if
	*/

	int* res = new int[array_size];

	int resnum;
	TIME_CPU(resnum = cpu_copy_if(res, src, array_size));


	int* host_result1 = (int*)malloc(array_size * sizeof(int));
	int* host_nres1 = (int*)malloc(sizeof(int));
	int* device_src;
	int* device_result1;
	int* nres1;
	CHECK(hipMalloc((void**)&device_src, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&device_result1, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&nres1, sizeof(int)));

	CHECK(hipMemcpy(device_src, src, array_size * sizeof(int), hipMemcpyHostToDevice));

	auto kernel_launcher = [&]() {
		warplevel_copy_if << < 512, 1024 >> > (device_result1, device_src, nres1, array_size);
		};

	TIME_GPU(kernel_launcher);

	CHECK(hipMemcpy(host_result1, device_result1, sizeof(int) * array_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_nres1, nres1, sizeof(int), hipMemcpyDeviceToHost));


	check_result(host_nres1, host_result1, &resnum, res);

	hipFree(device_src);
	hipFree(device_result1);
	hipFree(nres1);
	free(host_result1);
	free(host_nres1);
	free(res);
}