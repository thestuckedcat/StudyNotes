#include "cublas_example.h"
# include<stdio.h>
# include<stdlib.h>
# include<math.h>
# include<hip/hip_runtime.h>
# include"hipblas.h"


static __inline__ void modify(	hipblasHandle_t handle, 
								float* m, //data
								int ldm,  //��Ϊ������������������
								int n,	  //����
								int p,	  //������
								int q,	  //������
								float alpha, // scale1
								float beta) //scale2
{
	hipblasSscal(handle, n - q, &alpha, &m[IDX2C(p, q, ldm)], ldm);
	hipblasSscal(handle, ldm - p, &beta, &m[IDX2C(p, q, ldm)], 1);

}

int test_cublas() {
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	int i, j;
	float* devPtrA;

	float* a = 0;

	a = (float*)malloc(M * N * sizeof(*a));// ʹ��*a����sizeof(float)�����������ƶ�


	if (!a) {
		printf("host memory allocation failed\n");

		return EXIT_FAILURE;
		/*
			EXIT_FAILURE: �궨�壬ͨ��Ϊ1
			EXIT_SUCCESS���궨�壬ͨ��Ϊ0
		*/
	}

	for (j = 0; j < N; j++) {
		for (i = 0; i < M; i++) {
			a[IDX2C(i, j, M)] = (float)(1);
		}
	}

	cudaStat = hipMalloc((void**)&devPtrA, M * N * sizeof(*a));

	if (cudaStat != hipSuccess) {
		printf("Device memory allocation failed\n");
		return EXIT_FAILURE;
	}

	stat = hipblasCreate(&handle);

	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS initialization failed\n");
		return EXIT_FAILURE;
	}

	// �������ݵ�devPtrA
	stat = hipblasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("data download failed\n");
		hipFree(devPtrA);
		hipblasDestroy(handle);
		return EXIT_FAILURE;
	}


	modify(handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);
	stat = hipblasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);

	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("data upload failed");
		hipFree(devPtrA);
		hipblasDestroy(handle);
		return EXIT_FAILURE;
	}

	hipFree(devPtrA);
	hipblasDestroy(handle);
	for (j = 0; j < N; j++) {
		for (i = 0; i < M; i++) {
			printf("%7.0f", a[IDX2C(i, j, M)]);
		}
		printf("\n");
	}
	free(a);
	return EXIT_SUCCESS;







}