#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
# include "copy_if.h"
#include <stdio.h>
# include<random>
# include<iostream>
# include<chrono>
#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)


#define TIME_GPU(kernellauncher)	\
do						\
{						\
	hipEvent_t start, stop;\
	hipEventCreate(&start);\
	hipEventCreate(&stop);\
	hipEventRecord(start);\
	kernellauncher();\
	hipEventRecord(stop);\
	hipEventSynchronize(stop);\
	float milliseconds = 0;\
	hipEventElapsedTime(&milliseconds,start, stop);\
	std::cout << "GPU Time taken: " << milliseconds / 1000 << " seconds" << std::endl;\
	hipEventDestroy(start);\
	hipEventDestroy(stop);\
} while (0)

#define TIME_CPU(func_call)                             \
do {                                                    \
    auto start = std::chrono::high_resolution_clock::now(); \
    func_call;                                          \
    auto stop = std::chrono::high_resolution_clock::now();  \
    std::chrono::duration<double> duration = stop - start;  \
    std::cout << "CPU Time taken: " << duration.count() << " seconds" << std::endl; \
} while (0)




const int array_size = std::numeric_limits<int>::max()/2;//-1太容易出现问题了，例如循环溢出

void check_result(int* device_num, int* device_result, int* cpu_num, int* cpu_result) {
	if (*device_num != *cpu_num) {
		std::cout << "Device result is something wrong with cpu_result" << std::endl;
		std::cout << "device_num = " << *device_num << std::endl;
		std::cout << "cpu_num = " << *cpu_num << std::endl;

	}
	else {
		std::cout << "Device result is as same as cpu_result" << std::endl;
		/*
		int num = 0;
		for (int i = 0; i < *device_num; i++) {
			if (device_result[i] != cpu_result[i]) {
				num++;
				std::cout << "i-th result is not same where GPU is " << device_result[i] << " and CPU is " << cpu_result[i] << std::endl;
			}
		}
		std::cout << "The total no-match result num is " << num << std::endl;
		*/
	}
}

int* generate_random_array() {
	int* src = new int[array_size];


	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> distrib(-100, 100);
	for (int i = 0; i < array_size; i++) {
		src[i] = distrib(gen);
		//std::cout << src[i] << " ";
	}
	return src;
}

void compare1(int* src) {
	/*
	CPU vs original GPU copy_if
	*/
	int* res = new int[array_size];

	int resnum;
	TIME_CPU(resnum = cpu_copy_if(res, src, array_size));


	int* host_result1 = (int*)malloc(array_size * sizeof(int));
	int* host_nres1 = (int*)malloc(sizeof(int));
	int* device_src;
	int* device_result1;
	int* nres1;
	CHECK(hipMalloc((void**)&device_src, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&device_result1, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&nres1, sizeof(int)));

	CHECK(hipMemcpy(device_src, src, array_size * sizeof(int), hipMemcpyHostToDevice));
	
	auto kernel_launcher = [&](){
		origin_copy_if << < 512,1024 >> >(device_result1, device_src, nres1, array_size);
	};

	TIME_GPU(kernel_launcher);

	CHECK(hipMemcpy(host_result1, device_result1, sizeof(int) * array_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_nres1, nres1, sizeof(int), hipMemcpyDeviceToHost));


	check_result(host_nres1, host_result1, &resnum, res);

	hipFree(device_src);
	hipFree(device_result1);
	hipFree(nres1);
	free(host_result1);
	free(host_nres1);
	free(res);
}

void compare2(int* src) {
	/*
	CPU vs original GPU copy_if
	*/

	int* res = new int[array_size];

	int resnum;
	TIME_CPU(resnum = cpu_copy_if(res, src, array_size));


	int* host_result1 = (int*)malloc(array_size * sizeof(int));
	int* host_nres1 = (int*)malloc(sizeof(int));
	int* device_src;
	int* device_result1;
	int* nres1;
	CHECK(hipMalloc((void**)&device_src, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&device_result1, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&nres1, sizeof(int)));

	CHECK(hipMemcpy(device_src, src, array_size * sizeof(int), hipMemcpyHostToDevice));

	auto kernel_launcher = [&]() {
		blocklevel_copy_if << < 512,1024 >> > (device_result1, device_src, nres1, array_size);
		};

	TIME_GPU(kernel_launcher);

	CHECK(hipMemcpy(host_result1, device_result1, sizeof(int) * array_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_nres1, nres1, sizeof(int), hipMemcpyDeviceToHost));


	check_result(host_nres1, host_result1, &resnum, res);

	hipFree(device_src);
	hipFree(device_result1);
	hipFree(nres1);
	free(host_result1);
	free(host_nres1);
	free(res);
}

void compare3(int* src) {
	/*
	CPU vs original GPU copy_if
	*/

	int* res = new int[array_size];

	int resnum;
	TIME_CPU(resnum = cpu_copy_if(res, src, array_size));


	int* host_result1 = (int*)malloc(array_size * sizeof(int));
	int* host_nres1 = (int*)malloc(sizeof(int));
	int* device_src;
	int* device_result1;
	int* nres1;
	CHECK(hipMalloc((void**)&device_src, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&device_result1, array_size * sizeof(int)));
	CHECK(hipMalloc((void**)&nres1, sizeof(int)));

	CHECK(hipMemcpy(device_src, src, array_size * sizeof(int), hipMemcpyHostToDevice));

	auto kernel_launcher = [&]() {
		warplevel_copy_if << < 512,1024 >> > (device_result1, device_src, nres1, array_size);
		};

	TIME_GPU(kernel_launcher);

	CHECK(hipMemcpy(host_result1, device_result1, sizeof(int) * array_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_nres1, nres1, sizeof(int), hipMemcpyDeviceToHost));


	check_result(host_nres1, host_result1, &resnum, res);

	hipFree(device_src);
	hipFree(device_result1);
	hipFree(nres1);
	free(host_result1);
	free(host_nres1);
	free(res);
}

int main()
{
	int* src = generate_random_array();
	compare1(src);
	compare2(src);
	compare3(src);
	delete[] src;



}